#include "hip/hip_runtime.h"
#include <omp.h>
#include <iostream>
#include <cmath>
#include <chrono>
#include "libs\times.cpp"
#include "libs\ImageUtils.cu"
#include "libs\operators.cu"
#include "libs\mipchain.cu"

void GenerateMipMap(const float4 *inputImage, float4 *outputImage, const int2 isize, const int2 osize){
    const int fwidth = 2;
    const int fsize = (fwidth * fwidth);
    const float fweight = 1.0 / (float) fsize;

    #pragma omp parallel for
    for(int i = 0; i < osize.x * osize.y; i++){     
        
        int y = i / osize.x;
        int x = i % osize.x;

        //Dobbiamo invertire la condizione perchè in OpenMP
        //non è permesso avere un return dentro una sezione parallela
        if(y < osize.y || x < osize.x) {
            float4 result = make_float4(0, 0, 0, 0);
            for(int row = 0; row < fwidth; ++row){
                for(int column = 0; column < fwidth; ++column){
                    int2 boxOffset = make_int2(fwidth * x, fwidth * y);
                    int2 boxCoords = make_int2(boxOffset.x + row, boxOffset.y + column);

                    boxCoords.x = max(boxCoords.x, 0);
                    boxCoords.y = max(boxCoords.y, 0);
                    boxCoords.x = min(boxCoords.x, isize.x - 1);
                    boxCoords.y = min(boxCoords.y, isize.y - 1);
        
                    result = result + inputImage[tolinear(boxCoords, isize.x)];
                }
            }   
            outputImage[tolinear(make_int2(x, y), osize.x)] = result * fweight;
        }
    }


}

void generateMipMapChain(const std::string folder, const std::string destination, const std::string filename, times& measurements){

    const std::string filepath = folder + filename;

    auto start = std::chrono::steady_clock::now();
    auto original = ImageUtils::load(filepath.c_str());
    auto end = std::chrono::steady_clock::now();
    measurements.image_reading = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();
    measurements.image_reading /= 1000;

    auto containers = generateImageContainersForMipMaps(original);
    
    start = std::chrono::steady_clock::now();
    for(int i = 0; i < containers.size() - 1; ++i){
        int2 isize = make_int2(containers[i].width(), containers[i].height());
        int2 osize = make_int2(containers[i + 1].width(), containers[i + 1].height());
        GenerateMipMap((float4*) containers[i].raw_data(), (float4*) containers[i + 1].raw_data(), isize, osize);
    }
    end = std::chrono::steady_clock::now();
    measurements.image_processing = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();
    measurements.image_processing /= 1000;

    start = std::chrono::steady_clock::now();
    int writer_threads = containers.size() - 1;
    #pragma omp parallel for num_threads(writer_threads)
    for(int i = 1; i < containers.size(); i++){
        std::string save_dest = destination + "mip_" + std::to_string(containers[i].width()) + "x" + std::to_string(containers[i].height()) + "_" + filename;
        ImageUtils::save(containers[i], save_dest.c_str());
    }
    end = std::chrono::steady_clock::now();
    measurements.image_writing = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count(); 
    measurements.image_writing /= 1000;
}

int main(int argc, char** argv){

    const std::string filepath = std::string(argv[1]) + std::string(argv[3]);
    
    auto img = ImageUtils::load(filepath.c_str());

    std::cout << "algorithm" << "," << "image" << "," << "width" << "," << "height" << "," << "reading" << "," << "processing" << "," << "writing" << "\n";
    int runs = 10;
    for(int run = 1; run <= runs; run++){        
        times times;
        generateMipMapChain(std::string(argv[1]), std::string(argv[2]), std::string(argv[3]), times);
        std::cout 
        << "OpenMP" << ", " 
        << argv[3] << " , " 
        << img.width() << " , " << img.height() << " , " 
        << times.image_reading << ", " 
        << times.image_processing << ", "
        << times.image_writing << ""<< "\n";
    }


    return 0;
}
